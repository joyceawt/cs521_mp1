#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../include/utils.h"

#define NUM_RUNS 10
#define TILE_WIDTH 16  // also block size
#define BLOCK_SIZE 32

#define CUDA_CHECK(func)                                                   \
  do {                                                                     \
    hipError_t status = (func);                                           \
    if (status != hipSuccess) {                                           \
      printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__, \
             hipGetErrorString(status), status);                          \
      exit(EXIT_FAILURE);                                                  \
    }                                                                      \
  } while (0)

#define CHECK(name)                                                           \
  float *d_Aref_##name, *d_Bref_##name, *d_Cref_##name;                       \
  std::cerr << "checking " << #name << std::endl;                             \
  CUDA_CHECK(hipMalloc(&d_Aref_##name, Ref::M* Ref::K * sizeof(float)));     \
  CUDA_CHECK(hipMalloc(&d_Bref_##name, Ref::K* Ref::N * sizeof(float)));     \
  CUDA_CHECK(hipMalloc(&d_Cref_##name, Ref::M* Ref::N * sizeof(float)));     \
  CUDA_CHECK(hipMemcpy(d_Aref_##name, ref.A, Ref::M* Ref::K * sizeof(float), \
                        hipMemcpyHostToDevice));                             \
  CUDA_CHECK(hipMemcpy(d_Bref_##name, ref.B, Ref::K* Ref::N * sizeof(float), \
                        hipMemcpyHostToDevice));                             \
  float* d_Cref_INI_##name = new float[M * N]();                              \
  for (int i = 0; i < Ref::M; i++) {                                          \
    for (int j = 0; j < Ref::N; j++) {                                        \
      d_Cref_INI_##name[i * Ref::N + j] = 0;                                  \
    }                                                                         \
  }                                                                           \
  CUDA_CHECK(hipMemcpy(d_Cref_##name, d_Cref_INI_##name,                     \
                        Ref::M* Ref::N * sizeof(float),                       \
                        hipMemcpyHostToDevice));                             \
  name(d_Aref_##name, d_Bref_##name, d_Cref_##name, Ref::M, Ref::N, Ref::K);  \
  hipError_t err_c_##name = hipGetLastError();                              \
  if (err_c_##name != hipSuccess) {                                          \
    std::cerr << "CUDA Error: " << hipGetErrorString(err_c_##name)           \
              << std::endl;                                                   \
  }                                                                           \
  CUDA_CHECK(hipMemcpy(refC, d_Cref_##name, Ref::M* Ref::N * sizeof(float),  \
                        hipMemcpyDeviceToHost));                             \
  if (!ref.checkRef(refC)) {                                                  \
    std::cerr << "check ref failed!" << std::endl;                            \
  };

#define TIME(name)                                                          \
  float *d_A_##name, *d_B_##name, *d_C_##name;                              \
  CUDA_CHECK(hipMalloc(&d_A_##name, M* K * sizeof(float)));                \
  CUDA_CHECK(hipMalloc(&d_B_##name, K* N * sizeof(float)));                \
  CUDA_CHECK(hipMalloc(&d_C_##name, M* N * sizeof(float)));                \
  CUDA_CHECK(hipMemcpy(d_A_##name, A, M* K * sizeof(float),                \
                        hipMemcpyHostToDevice));                           \
  CUDA_CHECK(hipMemcpy(d_B_##name, B, K* N * sizeof(float),                \
                        hipMemcpyHostToDevice));                           \
  hipEvent_t start_##name, end_##name;                                     \
  hipEventCreate(&start_##name);                                           \
  hipEventCreate(&end_##name);                                             \
  float* d_C_INI_##name = new float[M * N]();                               \
  for (int i = 0; i < M; i++) {                                             \
    for (int j = 0; j < N; j++) {                                           \
      d_C_INI_##name[i * N + j] = 0;                                        \
    }                                                                       \
  }                                                                         \
  for (int i = 0; i < 2; i++) {                                             \
    CUDA_CHECK(hipMemcpy(d_C_##name, d_C_INI_##name, M* N * sizeof(float), \
                          hipMemcpyHostToDevice));                         \
    name(d_A_##name, d_B_##name, d_C_##name, M, N, K);                      \
  }                                                                         \
  hipError_t err_t_##name = hipGetLastError();                            \
  if (err_t_##name != hipSuccess) {                                        \
    std::cerr << "CUDA Error: " << hipGetErrorString(err_t_##name)         \
              << std::endl;                                                 \
  }                                                                         \
  float milliseconds_##name = 0;                                            \
  for (int i = 0; i < NUM_RUNS; i++) {                                      \
    CUDA_CHECK(hipMemcpy(d_C_##name, d_C_INI_##name, M* N * sizeof(float), \
                          hipMemcpyHostToDevice));                         \
    hipDeviceSynchronize();                                                \
    hipEventRecord(start_##name);                                          \
    name(d_A_##name, d_B_##name, d_C_##name, M, N, K);                      \
    hipEventRecord(end_##name);                                            \
    hipEventSynchronize(end_##name);                                       \
    float milliseconds_##i = 0;                                             \
    hipEventElapsedTime(&milliseconds_##i, start_##name, end_##name);      \
    milliseconds_##name += milliseconds_##i;                                \
  }                                                                         \
  hipMemcpy(C, d_C_##name, M* N * sizeof(float), hipMemcpyDeviceToHost);  \
  std::cout << "Time taken for GEMM (GPU, " << #name                        \
            << "): " << milliseconds_##name / (float)NUM_RUNS << "ms"       \
            << std::endl;                                                   \
  hipFree(d_A_##name);                                                     \
  hipFree(d_B_##name);                                                     \
  hipFree(d_C_##name);

__global__ void gemm_gpu_o0_kernel(float* A, float* B, float* C, int M, int N,
                                   int K) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    for (int i = 0; i < M; i++) {
      for (int j = 0; j < N; j++) {
        for (int k = 0; k < K; k++) {
          C[i * N + j] += A[i * K + k] * B[k * N + j];
        }
      }
    }
  }
}

void gemm_gpu_o0(float* A, float* B, float* C, int M, int N, int K) {
  // Init block and grid size
  dim3 blockSize(1);
  dim3 gridSize(1);
  gemm_gpu_o0_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// The scafolding for optimized GEMM implementations
__global__ void gemm_gpu_o1_kernel(float* A, float* B, float* C, int M, int N,
                                   int K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < N) {
    float sum = 0.0f;

    for (int k = 0; k < K; k++) {
      sum += A[row * K + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
  }
}
void gemm_gpu_o1(float* A, float* B, float* C, int M, int N,
                 int K) {  // Init block and grid size
  dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
  dim3 gridSize((N + blockSize.x - 1) / blockSize.x,
                (M + blockSize.y - 1) / blockSize.y);
  gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

__global__ void gemm_gpu_o2_kernel(float* A, float* B, float* C, int M, int N,
                                   int K) {
  // Initialized shared memory array As and Bs to store the sub-matrix of A and
  // B
  __shared__ float As[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

  int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

  float sum = 0.0f;

  // Loop over all the sub-matrices of A and B required to compute the block
  // sub-matrix (K sub-matrices/dimension)
  int numTiles = (K + TILE_WIDTH - 1) / TILE_WIDTH;  // handle partial tiles
  for (int t = 0; t < numTiles; t++) {
    // A sub-block; Load one tile of A into shared memory (if in bounds)
    int kA = t * TILE_WIDTH + threadIdx.x;  // column index of A
    if (row < M && kA < K) {
      As[threadIdx.y][threadIdx.x] = A[row * K + kA];
    } else {
      As[threadIdx.y][threadIdx.x] = 0.0f;  // if out of bounds, set to 0
    }

    // B sub-block; Load one tile of B into shared memory (if in bounds)
    int kB = t * TILE_WIDTH + threadIdx.y;  // row index of B
    if (kB < K && col < N) {
      Bs[threadIdx.y][threadIdx.x] = B[kB * N + col];
    } else {
      Bs[threadIdx.y][threadIdx.x] = 0.0f;
    }

    __syncthreads();

    // Multiply the two sub-matrices for this tile
    for (int i = 0; i < TILE_WIDTH; i++) {
      sum += As[threadIdx.y][i] * Bs[i][threadIdx.x];
    };

    __syncthreads();
  }

  if (row < M && col < N) {
    C[row * N + col] = sum;
  }
}

void gemm_gpu_o2(float* A, float* B, float* C, int M, int N, int K) {
  // Init block and grid size
  dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
  dim3 gridSize((N + TILE_WIDTH - 1) / TILE_WIDTH,
                (M + TILE_WIDTH - 1) / TILE_WIDTH);
  gemm_gpu_o2_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

__global__ void gemm_gpu_o3_kernel(float* A, float* B, float* C, int M, int N,
                                   int K) {
  // Initialized shared memory array As and Bs to store the sub-matrix of A and
  // B
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

  int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  float sum = 0.0f;

  // Loop over all the sub-matrices of A and B required to compute the block
  // sub-matrix (K sub-matrices/dimension)
  int numTiles = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;  // handle partial tiles
  for (int t = 0; t < numTiles; t++) {
    // A sub-block; Load one tile of A into shared memory (if in bounds)
    int kA = t * BLOCK_SIZE + threadIdx.x;  // column index of A
    if (row < M && kA < K) {
      As[threadIdx.y][threadIdx.x] = A[row * K + kA];
    } else {
      As[threadIdx.y][threadIdx.x] = 0.0f;  // if out of bounds, set to 0
    }

    // B sub-block; Load one tile of B into shared memory (if in bounds)
    int kB = t * BLOCK_SIZE + threadIdx.y;  // row index of B
    if (kB < K && col < N) {
      Bs[threadIdx.y][threadIdx.x] = B[kB * N + col];
    } else {
      Bs[threadIdx.y][threadIdx.x] = 0.0f;
    }

    __syncthreads();

    // Multiply the two sub-matrices for this tile
    for (int i = 0; i < BLOCK_SIZE; i++) {
      sum += As[threadIdx.y][i] * Bs[i][threadIdx.x];
    };

    __syncthreads();
  }

  if (row < M && col < N) {
    C[row * N + col] = sum;
  }
}
void gemm_gpu_o3(float* A, float* B, float* C, int M, int N, int K) {
  // Init block and grid size// Init block and grid size
  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
  gemm_gpu_o3_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

int main(int argc, char* argv[]) {
  if (argc < 3) {
    std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
    return 1;
  }

  int M = atoi(argv[1]);
  int N = atoi(argv[2]);
  int K = atoi(argv[3]);

  // int runs = atoi(argv[3]);
  float* A = new float[M * K]();
  float* B = new float[K * N]();
  float* C = new float[M * N]();

  fillRandom(A, M * K);
  fillRandom(B, K * N);

  /// GPU Implementation
  // Check if implementation is correct
  auto ref = Ref();
  float* refC = new float[Ref::M * Ref::N]();
  CHECK(gemm_gpu_o0)
  CHECK(gemm_gpu_o1)
  CHECK(gemm_gpu_o2)
  CHECK(gemm_gpu_o3)

  // Actual run
  TIME(gemm_gpu_o0)
  TIME(gemm_gpu_o1)
  TIME(gemm_gpu_o2)
  TIME(gemm_gpu_o3)

  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);

  delete[] A;
  delete[] B;
  delete[] C;

  return 0;
}